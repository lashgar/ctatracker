
#include <hip/hip_runtime.h>
__global__ void executeThirdLayer(float *Layer3_Neurons_GPU, float *Layer3_Weights_GPU,float *Layer4_Neurons_GPU)
{
	int blockID=blockIdx.x;
	//int pixelY=threadIdx.y;


	int weightBegin=blockID*1251;
 
	float result=0;

	result+=Layer3_Weights_GPU[weightBegin];

	++weightBegin;

    for (int i=0; i<1250; ++i )
    {
		result+=Layer3_Neurons_GPU[i+(1250*blockIdx.y)]*Layer3_Weights_GPU[weightBegin+i];
    }

	result=(1.7159*tanhf(0.66666667*result));

	Layer4_Neurons_GPU[blockID+(100*blockIdx.y)]=result;

}

