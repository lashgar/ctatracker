#include "hip/hip_runtime.h"
__global__ void Pathcalc_Portfolio_KernelGPU(float *d_v, float *d_Lb)
{
  const int     tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int threadN = blockDim.x * gridDim.x;

  int   i,path;
  float L[NN], L2[L2_SIZE], z[NN];
  float *L_b = L;
  
  /* Monte Carlo LIBOR path calculation*/

  for(path = tid; path < NPATH; path += threadN){
    // initialise the data for current thread
    for (i=0; i<N; i++) {
      // for real application, z should be randomly generated
      z[i] = 0.3;
      L[i] = 0.05;
    }
    path_calc_b1(L, z, L2);
    d_v[path] = portfolio_b(L,L_b);
    path_calc_b2(L_b, z, L2);
    d_Lb[path] = L_b[NN-1];
  }
}


