
#include <hip/hip_runtime.h>
__global__ 
void euclid(char *data, float x2, float y2,float *z, int N, int W, int L_POS)
{
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	float tmp_lat=0.0, tmp_long=0.0;
	int position = ( idx * W ) + L_POS - 1;	
	
	if(idx < N) {
		char temp1[5];
		for( int i = 0 ; i < 5 ; i++ ) {
			temp1[i] = data[position+i];
		}
		char temp2[5];
		for( int i = 0 ; i < 5 ; i++ ) {
			temp2[i] = data[position+6+i];
		}
		
		int dig1, dig2, dig3, dig_1;
		if( temp1[0] == ' ' ) { dig1 = 0; }
		else {
			dig1 = temp1[0] - 48;
			tmp_lat += dig1 * 100;
		}
		if( temp1[1] == ' ' ) { dig2 = 0; }
		else {
			dig2 = temp1[1] - 48;
			tmp_lat += dig2 * 10;
		}
		if( temp1[2] == ' ' ) { dig3 = 0; }
		else {
			dig3 = temp1[2] - 48;
			tmp_lat += dig3 * 1;
		}
		dig_1 = temp1[4] - 48;
		tmp_lat += (float) dig_1 / 10;

		if( temp2[0] == ' ' ) { dig1 = 0; }
		else {
			dig1 = temp2[0] - 48;
			tmp_long += dig1 * 100;
		}
		if( temp2[1] == ' ' ) { dig2 = 0; }
		else {
			dig2 = temp2[1] - 48;
			tmp_long += dig2 * 10;
		}
		if( temp2[2] == ' ' ) { dig3 = 0; }
		else {
			dig3 = temp2[2] - 48;
			tmp_long += dig3 * 1;
		}
		dig_1 = temp2[4] - 48;
		tmp_long += (float) dig_1 / 10;

		z[idx]=sqrt(((tmp_lat-x2)*(tmp_lat-x2))+((tmp_long-y2)*(tmp_long-y2)));
	}
}


