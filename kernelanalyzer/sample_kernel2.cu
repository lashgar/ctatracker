#include "hip/hip_runtime.h"
template <class T, unsigned int blockSize>
__global__ void
FUNC(reduce5)(T *g_idata, T *g_odata, unsigned int n)
{
    SharedMemory<T> smem;
    T *sdata = smem.getPointer();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int j = blockIdx.x*(blockSize*2) + threadIdx.x, k = threadIdx.x;
  int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

    sdata[tid] = (i < n) ? g_idata[i] : 0;
    if (i + blockSize < n) 
        sdata[tid] += g_idata[i+blockSize];  

}


