#include "hip/hip_runtime.h"
__global__ void fwtBatch2Kernel(
    float *d_Output,
    float *d_Input,
    int stride){
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int   N = blockDim.x *  gridDim.x * 4;
    const int offset=blockIdx.y * N;

    //float *d_Src = d_Input  + blockIdx.y * N;
    //float *d_Dst = d_Output + blockIdx.y * N;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    float D0 = d_Input[offset+i0];
    float D1 = d_Input[offset+i1];
    float D2 = d_Input[offset+i2];
    float D3 = d_Input[offset+i3];

    float T;
    T = D0; D0        = D0 + D2; D2        = T - D2;
    T = D1; D1        = D1 + D3; D3        = T - D3;
    T = D0; d_Output[offset+i0] = D0 + D1; d_output[offset+i1] = T - D1;
    T = D2; d_Output[offset+i2] = D2 + D3; d_output[offset+i3] = T - D3;
}
