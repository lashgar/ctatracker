#include "hip/hip_runtime.h"
__global__ void 
findK(	long height,
		knode *knodesD,
		long knodes_elem,
		record *recordsD,

		long *currKnodeD,
		long *offsetD,
		int *keysD, 
		record *ansD)
{

	// private thread IDs
	int thid = threadIdx.x;
	int bid = blockIdx.x;

	// processtree levels
	int i;
	for(i = 0; i < height; i++){

		if((knodesD[currKnodeD[bid]].keys[thid]) <= keysD[bid] && (knodesD[currKnodeD[bid]].keys[thid+1] > keysD[bid])){
			if(knodesD[offsetD[bid]].indices[thid] < knodes_elem){
				offsetD[bid] = knodesD[offsetD[bid]].indices[thid];
			}
		}
		__syncthreads();

		// set for next tree level
		if(thid==0){
			currKnodeD[bid] = offsetD[bid];
		}
		__syncthreads();

	}

	if(knodesD[currKnodeD[bid]].keys[thid] == keysD[bid]){
		ansD[bid].value = recordsD[knodesD[currKnodeD[bid]].indices[thid]].value;
	}

}
