
#include <hip/hip_runtime.h>
__global__ void executeFourthLayer(float *Layer4_Neurons_GPU,float *Layer4_Weights_GPU,float *Layer5_Neurons_GPU)
{
	int blockID=blockIdx.x;
	//int pixelY=threadIdx.y;


	int weightBegin=blockID*101;
 
	float result=0;

	result+=Layer4_Weights_GPU[weightBegin];

	++weightBegin;

    for (int i=0; i<100; ++i )
    {
		result+=Layer4_Neurons_GPU[i+(100*blockIdx.y)]*Layer4_Weights_GPU[weightBegin+i];
    }

	result=(1.7159*tanhf(0.66666667*result));

	Layer5_Neurons_GPU[blockID+(10*blockIdx.y)]=result;
}
