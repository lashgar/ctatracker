#include "hip/hip_runtime.h"
__global__  __device__ void render(float4* d_debug_float4, uint* d_debug_uint, uint * result, Node * dnode, uint imageW, uint imageH, float pas, float df)
{
	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	uint tid(__umul24(threadIdx.y, blockDim.x) + threadIdx.x);

	uint id=(x + y * imageW);
	float4 pile[5];
	uint Obj, nRec(5), n(0);
	float prof, tmp;

	for( int i(0); i < nRec; ++i )
		pile[i] = make_float4(0.0f,0.0f,0.0f,1.0f);

	if( x < imageW && y < imageH )
	{
		prof = 10000.0f;
		result[id] = 0;
		float tPixel(2.0f/float(min(imageW,imageH)));
		float4 f(make_float4(0.0f,0.0f,0.0f,1.0f));
		matrice3x4 M(MView);
		Rayon R;
		R.A = make_float3(M.m[0].w,M.m[1].w,M.m[2].w);
		R.u = make_float3(M.m[0])*df
			+ make_float3(M.m[2])*(float(x)-float(imageW)*0.5f)*tPixel
			+ make_float3(M.m[1])*(float(y)-float(imageH)*0.5f)*tPixel;
		R.u = normalize(R.u);
		__syncthreads();

		for( int i(0); i < nRec && n == i; i++ ) {

			for( int j(0); j < numObj; j++ ) {
				Node nod(cnode[j]);
				Sphere s(nod.s);
				float t;
				s.C.x += pas;
				if( nod.fg )
					t = intersectionPlan(R,s.C,s.C);
				else
					t = intersectionSphere(R,s.C,s.r);

				if( t > 0.0f && t < prof ) {
					prof = t;
					Obj = j;
				}
			}
			float t = prof;
			if( t > 0.0f && t < 10000.0f ) {
				n++;
				Node nod(cnode[Obj]);
				Sphere s(nod.s);
				s.C.x += pas;
				float4 color(make_float4(s.R,s.V,s.B,s.A));
				float3 P(R.A+R.u*t), L(normalize(make_float3(10.0f,10.0f,10.0f)-P)), V(normalize(R.A-P));
				float3 N(nod.fg?getNormaleP(P):getNormale(P,s.C));
				float3 Np(dot(V,N)<0.0f?(-1*N):N);
				pile[i] = 0.05f * color;
            if( dot(Np,L) > 0.0f && notShadowRay(cnode,P,L,pas) ) {
					float3 Ri(normalize(L+V));
					//Ri = (L+V)/normalize(L+V);
					pile[i] += 0.3f * color* (min(1.0f,dot(Np,L)));

               #ifdef FIXED_CONST_PARSE
					tmp = 0.8f * pow(max(0.0f,min(1.0f,dot(Np,Ri))),50.0f);
               #else
               tmp = 0.8f * float2int_pow50(max(0.0f,min(1.0f,dot(Np,Ri))));
               #endif
					pile[i].x += tmp;
					pile[i].y += tmp;
					pile[i].z += tmp;

				}

				R.u = 2.0f*N*dot(N,V) - V;
				R.u = normalize(R.u);
				R.A = P+R.u*0.0001f;
			}
			prof = 10000.0f;
		}
      for( int i(n-1); i > 0; i-- )
				pile[i-1] = pile[i-1] + 0.8f*pile[i];
      result[id] += rgbaFloatToInt(pile[0]);
	}
}

