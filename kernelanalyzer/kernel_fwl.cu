
#include <hip/hip_runtime.h>
__global__ void fwtBatch2Kernel(
    float *d_Output,
    float *d_Input,
    int stride
){
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int   N = blockDim.x *  gridDim.x * 4;

    float *d_Src = d_Input  + blockIdx.y * N;
    float *d_Dst = d_Output + blockIdx.y * N;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    float D0 = d_Src[i0];
    float D1 = d_Src[i1];
    float D2 = d_Src[i2];
    float D3 = d_Src[i3];

    float T;
    T = D0; D0        = D0 + D2; D2        = T - D2;
    T = D1; D1        = D1 + D3; D3        = T - D3;
    T = D0; d_Dst[i0] = D0 + D1; d_Dst[i1] = T - D1;
    T = D2; d_Dst[i2] = D2 + D3; d_Dst[i3] = T - D3;
}


