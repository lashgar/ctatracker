#include "hip/hip_runtime.h"
__global__ void sobel_gpu(QUANTUM_TYPE *img_out, QUANTUM_TYPE *img_in, int WIDTH, int HEIGHT){
	int xc,yc;
	xc=blockDim.x*blockIdx.x+threadIdx.x;
	yc=blockDim.y*blockIdx.y+threadIdx.y;
    QUANTUM_TYPE LUp,LCnt,LDw,RUp,RCnt,RDw;
	int pixel;
    int xm1=xc-1;
    int xp1=xc+1;
    int ym1=yc-1;
    int yp1=yc+1;


	if(xc<WIDTH && yc<HEIGHT){
        QUANTUM_TYPE p1=img_in[xm1+ym1*WIDTH];
        QUANTUM_TYPE p2=img_in[xm1+yc*WIDTH];
        QUANTUM_TYPE p3=img_in[xm1+yp1*WIDTH];
        QUANTUM_TYPE p4=img_in[xp1+ym1*WIDTH];
        QUANTUM_TYPE p5=img_in[xp1+yc*WIDTH];
        QUANTUM_TYPE p6=img_in[xp1+yp1*WIDTH];
        LUp = (xc-1>=0 && yc-1>=0)? p1:0;
        LCnt= (xc-1>=0)? p2:0;
        LDw = (xc-1>=0 && yc+1<HEIGHT)? p3:0;
        RUp = (xc+1<WIDTH && yc-1>=0)? p4:0;
        RCnt= (xc+1<WIDTH)? p5:0;
        RDw = (xc+1<WIDTH && yc+1<HEIGHT)? p6:0;
        pixel = -1*LUp  + 1*RUp +
                -2*LCnt + 2*RCnt +
                -1*LDw  + 1*RDw;
        pixel=(pixel<0)?0:pixel;
        pixel=(pixel>MAXRGB)?MAXRGB:pixel;
        img_out[xc+yc*WIDTH]=pixel;
	}
}
