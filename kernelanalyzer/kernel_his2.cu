#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// GPU-specific common definitions
////////////////////////////////////////////////////////////////////////////////
//Data type used for input data fetches
/*
typedef uint4 data_t;

//Both map to single instructions on G8x / G9x / G10x
#define UMUL(a, b)      __umul24( (a), (b) )
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

//May change on future hardware, so better parametrize the code
#define SHARED_MEMORY_BANKS 16

//Threadblock size: must be a multiple of (4 * SHARED_MEMORY_BANKS)
//because of the bit permutation of threadIdx.x
#define HISTOGRAM64_THREADBLOCK_SIZE (4 * SHARED_MEMORY_BANKS)

*/

////////////////////////////////////////////////////////////////////////////////
// Merge histogram64() output
// Run one threadblock per bin; each threadbock adds up the same bin counter 
// from every partial histogram. Reads are uncoalesced, but mergeHistogram64
// takes only a fraction of total processing time
////////////////////////////////////////////////////////////////////////////////
#define MERGE_THREADBLOCK_SIZE 256

__global__ void mergeHistogram64Kernel(
    uint *d_Histogram,
    uint *d_PartialHistograms,
    uint histogramCount
){
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];

    uint sum = 0;
    for(uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
    data[threadIdx.x] = sum;

    for(uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1){
        __syncthreads();
        if(threadIdx.x < stride)
            data[threadIdx.x] += data[threadIdx.x + stride];
    }

    if(threadIdx.x == 0)
        d_Histogram[blockIdx.x] = data[0];
}
