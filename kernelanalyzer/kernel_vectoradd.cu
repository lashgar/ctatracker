#include "hip/hip_runtime.h"
__global__ void
Kernel( float *a, float *b, float *c)
{
    int l = a[tid];
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
    a[tid] = b[tid] + c[tid];
}
