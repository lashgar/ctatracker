#include "hip/hip_runtime.h"
__global__ void Pathcalc_Portfolio_KernelGPU2(float *d_v)
{
  const int     tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int threadN = blockDim.x * gridDim.x;

  int   i, path;
  float L[NN], z[NN];
  
  /* Monte Carlo LIBOR path calculation*/

  for(path = tid; path < NPATH; path += threadN){
    // initialise the data for current thread
    for (i=0; i<N; i++) {
      // for real application, z should be randomly generated
      z[i] = 0.3;
      L[i] = 0.05;
    }	   
    path_calc(L, z);
    d_v[path] = portfolio(L);
  }
}


