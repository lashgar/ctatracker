#include "hip/hip_runtime.h"
__global__ void big_kernel(Cell *cells, int *cnumPars,Cell *cells2, int *cnumPars2,struct kernel_consts *dev,int *border) {

    int ix;
    int iy;
    int iz;

    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;
    int nz = blockDim.z * gridDim.z;

    ix = blockIdx.x * blockDim.x + threadIdx.x;
    iy = blockIdx.y * blockDim.y + threadIdx.y;
    iz = blockIdx.z * blockDim.z + threadIdx.z;

    //printf("x: %d : %d\n",nx,blockDim.x * gridDim.x);
    //printf("y: %d : %d\n",ny,blockDim.y * gridDim.y);
    //printf("z: %d : %d\n",nz,blockDim.z * gridDim.z);

    //move common declarations on top

    int index = (iz*ny + iy)*nx + ix;
    int np;  //internal loop limit

    //this should be moved to shared memory
    Cell &cell = cells[index];  //just a reference to the correspondig cell //FIXME

    int neighCells[27];

    //it is safe to move the call here, neighbours do not change between the two original calls

    //move this computation to cpu
    //const float tc_orig = hSq*hSq*hSq;

    const float parSize = 0.0002f;
    const float epsilon = 1e-10f;
    const float stiffness = 30000.f;
    const float damping = 128.f;

    /*
    for (i=0;i<27;i++) {
        neighCells[i] = 0xffffffff;
    }
    */
    int numNeighCells = InitNeighCellList(ix, iy, iz, neighCells,cnumPars);

    /*
    //printf("thread %d: number of neighbors: %d\n",index,numNeighCells);
    for (int i=0;i<numNeighCells;i++) {
        printf("thread %d : %d-th neighbor %d\n",index,i,neighCells[i]);
    }
    */

    ////////////////////////////////////////////////////////////////////////////////
    //void ClearParticlesMT(int i) {
    ////////////////////////////////////////////////////////////////////////////////

    /**/


    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //    for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //        for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    cnumPars[index] = 0;

    //                }  //close nested loop;



    __syncthreads();



    //} close ClearParticlesMT()
    ////////////////////////////////////////////////////////////////////////////////
    //void RebuildGridMT(int i) {




    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //        for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //            for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    Cell const &cell2 = cells2[index];
    int np2 = cnumPars2[index];

    for (int j = 0; j < np2; ++j) {
        int ci = (int)((cell2.p[j].x - domainMin.x) / dev->delta.x);
        int cj = (int)((cell2.p[j].y - domainMin.y) / dev->delta.y);
        int ck = (int)((cell2.p[j].z - domainMin.z) / dev->delta.z);

        if (ci < 0) ci = 0; else if (ci > (nx-1)) ci = nx-1;
        if (cj < 0) cj = 0; else if (cj > (ny-1)) cj = ny-1;
        if (ck < 0) ck = 0; else if (ck > (nz-1)) ck = nz-1;

        int index2 = (ck*ny + cj)*nx + ci;
        // this assumes that particles cannot travel more than one grid cell per time step
        int np_renamed = cnumPars[index2];

        if (border[index2]) {
            //use atomic
            atomicAdd(&cnumPars[index2],1);
        } else {
            cnumPars[index2]++;
        }

        //#warning what if we exceed CELL_PARTICLES particles per cell here??
        //from what I see is that we calculate the same frame over and over
        //so every cell has at most CELL_PARTICLES particles, from the initialisation


        Cell &cell_renamed = cells[index2];
        cell_renamed.p[np_renamed].x = cell2.p[j].x;
        cell_renamed.p[np_renamed].y = cell2.p[j].y;
        cell_renamed.p[np_renamed].z = cell2.p[j].z;
        cell_renamed.hv[np_renamed].x = cell2.hv[j].x;
        cell_renamed.hv[np_renamed].y = cell2.hv[j].y;
        cell_renamed.hv[np_renamed].z = cell2.hv[j].z;
        cell_renamed.v[np_renamed].x = cell2.v[j].x;
        cell_renamed.v[np_renamed].y = cell2.v[j].y;
        cell_renamed.v[np_renamed].z = cell2.v[j].z;
        //cell_renamed.debug[np_renamed] = index2;
    }

    //                }  //close nested loops



    __syncthreads();




    //} close RebuildGridMT()
    ////////////////////////////////////////////////////////////////////////////////
    //void InitDensitiesAndForcesMT(int i) {

    //from now on we don't change the cnumPars[index]
    np = cnumPars[index];  //internal loop limit


    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //        for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //            for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    //    Cell &cell = cells[index];

    //    int np = cnumPars[index];

    for (int j = 0; j < np; ++j) {
        cell.density[j] = 0.f;
        cell.a[j].x = externalAcceleration.x;
        cell.a[j].y = externalAcceleration.y;
        cell.a[j].z = externalAcceleration.z;
    }


    //                }  //close nested loops



    __syncthreads();




    //} close InitDensitiesAndForcesMT()
    ////////////////////////////////////////////////////////////////////////////////
    //void ComputeDensitiesMT(int i) {




    //    int neighCells[27];

    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //        for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //            for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    //    int np = cnumPars[index];

    //    if (np == 0)  continue;
    //
    // if np==0 we do net enter the following loop

    //    int numNeighCells = InitNeighCellList(ix, iy, iz, neighCells);

    //    Cell &cell = cells[index];

    Vec3 tmp;

    for (int j = 0; j < np; ++j)
        for (int inc = 0; inc < numNeighCells; ++inc) {
            int indexNeigh = neighCells[inc];
            Cell &neigh = cells[indexNeigh];
            int numNeighPars = cnumPars[indexNeigh];
            for (int iparNeigh = 0; iparNeigh < numNeighPars; ++iparNeigh)
                if (&neigh.p[iparNeigh] < &cell.p[j]) {
                    //float distSq = (cell.p[j] - neigh.p[iparNeigh]).GetLengthSq();
                    float distSq;
                    operator_sub(&tmp,&cell.p[j],&neigh.p[iparNeigh]);
                    distSq = GetLengthSq(&tmp);
                    if (distSq < dev->hSq) {
                        float t = dev->hSq - distSq;
                        float tc = t*t*t;

                        if (border[index]) {
                            //use atomic
                            atomicAdd(&cell.density[j],tc);
                        } else {
                            cell.density[j] += tc;
                        }

                        if (border[indexNeigh]) {
                            //use atomic
                            atomicAdd(&neigh.density[iparNeigh],tc);
                        } else {
                            neigh.density[iparNeigh] += tc;
                        }
                    }
                }
            ;
        }

    //                }  //close nested loops



    __syncthreads();


    //} close ComputeDensitiesMT()
    ////////////////////////////////////////////////////////////////////////////////
    //void ComputeDensities2MT(int i) {




    //    const float tc = hSq*hSq*hSq;


    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //        for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //            for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    //    Cell &cell = cells[index];

    //    int np = cnumPars[index];

    for (int j = 0; j < np; ++j) {
        cell.density[j] += dev->tc_orig;
        cell.density[j] *= dev->densityCoeff;
    }

    //                }  //close nested loops



    __syncthreads();




    //} close ComputeDensities2MT()
    ////////////////////////////////////////////////////////////////////////////////
    //void ComputeForcesMT(int i) {




    //    int neighCells[27];

    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //        for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //            for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    //    int np = cnumPars[index];

    //    if (np == 0)  continue;
    //
    // if np==0 we do net enter the following loop

    //    int numNeighCells = InitNeighCellList(ix, iy, iz, neighCells);

    //    Cell &cell = cells[index];

    for (int j = 0; j < np; ++j)
        for (int inc = 0; inc < numNeighCells; ++inc) {
            int indexNeigh = neighCells[inc];
            Cell &neigh = cells[indexNeigh];
            int numNeighPars = cnumPars[indexNeigh];
            for (int iparNeigh = 0; iparNeigh < numNeighPars; ++iparNeigh)
                if (&neigh.p[iparNeigh] < &cell.p[j]) {
                    //Vec3 disp = cell.p[j] - neigh.p[iparNeigh];
                    //float distSq = disp.GetLengthSq();
                    Vec3 disp;
                    operator_sub(&disp,&cell.p[j],&neigh.p[iparNeigh]);
                    float distSq = GetLengthSq(&disp);
                    if (distSq < dev->hSq) {
                        //float dist = sqrtf(std::max(distSq, 1e-12f));
                        float dist = sqrtf(fmax(distSq, 1e-12f));
                        float hmr = dev->h - dist;

                        //Vec3 acc = disp * pressureCoeff * (hmr*hmr/dist) *
                        //    (cell.density[j]+neigh.density[iparNeigh] - doubleRestDensity);

                        //acc += (neigh.v[iparNeigh] - cell.v[j]) * viscosityCoeff * hmr;
                        //acc /= cell.density[j] * neigh.density[iparNeigh];

                        Vec3 acc;
                        operator_mult(&acc,&disp, dev->pressureCoeff * (hmr*hmr/dist) *
                                      (cell.density[j]+neigh.density[iparNeigh] - doubleRestDensity));

                        operator_sub(&tmp,&neigh.v[iparNeigh],&cell.v[j]);
                        operator_mult(&tmp,&tmp,dev->viscosityCoeff * hmr);
                        operator_add(&acc,&acc,&tmp);
                        operator_div(&acc,&acc,cell.density[j] * neigh.density[iparNeigh]);

                        if (border[index]) {
                            //use atomics
#warning this works because no one reads these values at the moment ??
                            atomicAdd(&cell.a[j].x,acc.x);
                            atomicAdd(&cell.a[j].y,acc.y);
                            atomicAdd(&cell.a[j].z,acc.z);
                        } else {
                            operator_add(&cell.a[j],&cell.a[j],&acc);
                        }

                        if (border[indexNeigh]) {
                            //use atomics
#warning this works because no one reads these values at the moment ??
                            //reminder: there is no atomicSub for floats, so we add the negative value
                            atomicAdd(&neigh.a[iparNeigh].x,-acc.x);
                            atomicAdd(&neigh.a[iparNeigh].y,-acc.y);
                            atomicAdd(&neigh.a[iparNeigh].z,-acc.z);
                        } else {
                            operator_sub(&neigh.a[iparNeigh],&neigh.a[iparNeigh],&acc);
                        }
                    }
                }
        }

    //                }  //close nested loops



    __syncthreads();




    //} close ComputeForcesMT()
    ////////////////////////////////////////////////////////////////////////////////
    //void ProcessCollisionsMT(int i) {




    //    const float parSize = 0.0002f;
    //    const float epsilon = 1e-10f;
    //    const float stiffness = 30000.f;
    //    const float damping = 128.f;

    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //        for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //            for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    //    Cell &cell = cells[index];

    //    int np = cnumPars[index];

    for (int j = 0; j < np; ++j) {
        //Vec3 pos = cell.p[j] + cell.hv[j] * timeStep;
        Vec3 pos;
        operator_mult(&pos,&cell.hv[j],timeStep);
        operator_add(&pos,&pos,&cell.p[j]);

        float diff = parSize - (pos.x - domainMin.x);
        if (diff > epsilon)
            cell.a[j].x += stiffness*diff - damping*cell.v[j].x;

        diff = parSize - (domainMax.x - pos.x);
        if (diff > epsilon)
            cell.a[j].x -= stiffness*diff + damping*cell.v[j].x;

        diff = parSize - (pos.y - domainMin.y);
        if (diff > epsilon)
            cell.a[j].y += stiffness*diff - damping*cell.v[j].y;

        diff = parSize - (domainMax.y - pos.y);
        if (diff > epsilon)
            cell.a[j].y -= stiffness*diff + damping*cell.v[j].y;

        diff = parSize - (pos.z - domainMin.z);
        if (diff > epsilon)
            cell.a[j].z += stiffness*diff - damping*cell.v[j].z;

        diff = parSize - (domainMax.z - pos.z);
        if (diff > epsilon)
            cell.a[j].z -= stiffness*diff + damping*cell.v[j].z;
    }

    //                }  //close nested loops



    __syncthreads();




    //} close ProcessCollisionsMT()
    ////////////////////////////////////////////////////////////////////////////////
    //void AdvanceParticlesMT(int i) {




    //    for (int iz = grids[i].sz; iz < grids[i].ez; ++iz)
    //        for (int iy = grids[i].sy; iy < grids[i].ey; ++iy)
    //            for (int ix = grids[i].sx; ix < grids[i].ex; ++ix) {

    //    int index = (iz*ny + iy)*nx + ix;

    //    Cell &cell = cells[index];

    //    int np = cnumPars[index];

    for (int j = 0; j < np; ++j) {
        //Vec3 v_half = cell.hv[j] + cell.a[j]*timeStep;
        Vec3 v_half;
        operator_mult(&v_half,&cell.a[j],timeStep);
        operator_add(&v_half,&v_half,&cell.hv[j]);

        //cell.hv[j] = v_half;
        cell.hv[j].x = v_half.x;
        cell.hv[j].y = v_half.y;
        cell.hv[j].z = v_half.z;

        //cell.v[j] *= 0.5f;
        operator_mult(&cell.v[j],&cell.v[j],0.5f);

        //cell.v[j] = cell.hv[j] + v_half;
        operator_add(&cell.v[j],&cell.hv[j],&v_half);

        //we can change v_half now, (we want to use only one tmp variable)
        //cell.p[j] += v_half * timeStep;
        operator_mult(&v_half,&v_half,timeStep);
        operator_add(&cell.p[j],&cell.p[j],&v_half);
    }

    //                }  //close nested loops



    __syncthreads();




    //} close AdvanceParticlesMT()
    ////////////////////////////////////////////////////////////////////////////////

/**/


} //close big_kernel()

