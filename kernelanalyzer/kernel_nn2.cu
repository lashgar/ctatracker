#include "hip/hip_runtime.h"

__global__ void executeSecondLayer(float *Layer2_Neurons_GPU, float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU)
{
	int blockID=blockIdx.x;
	int pixelX=threadIdx.x;
	int pixelY=threadIdx.y;


	int weightBegin=blockID*26*6;
	int windowX=pixelX*2;
	int windowY=pixelY*2;
    
	float result=0;

	
	result+=Layer2_Weights_GPU[weightBegin];
	
	if(blockID==1 && pixelX==0 && pixelY==0)
	{
		result+=0;
	}

	++weightBegin;

	for (int i=0; i<25; ++i )
    {
        result+=Layer2_Neurons_GPU[(windowX + 13*windowY +kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6];
        result+=Layer2_Neurons_GPU[(169 + windowX + 13*windowY +kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+1];
	result+=Layer2_Neurons_GPU[(338 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+2];
        result+=Layer2_Neurons_GPU[(507 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+3];
        result+=Layer2_Neurons_GPU[(676 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+4];
        result+=Layer2_Neurons_GPU[(845 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+5];
	}

	result=(1.7159*tanhf(0.66666667*result));

	Layer3_Neurons_GPU[(5*5*blockID+pixelY*5+pixelX)+(1250*blockIdx.y)]=result;
}

