#include "hip/hip_runtime.h"
__global__ void
srad_cuda_1(
		  float *E_C, 
		  float *W_C, 
		  float *N_C, 
		  float *S_C,
		  float * J_cuda, 
		  float * C_cuda, 
		  int cols, 
		  int rows, 
		  float q0sqr
)
{
  //int a=0,b=1,c=3;
  //block id
  int bx = blockIdx.x;
  int by = blockIdx.y;

  //thread id
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  //indices
  int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
  int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
  int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
  int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
  int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

  float n, w, e, s, jc, g2, l, num, den, qsqr, c;

  //shared memory allocation
  __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float temp_result[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ float north[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float south[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  east[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  west[BLOCK_SIZE][BLOCK_SIZE];

  //load data to shared memory
  north[ty][tx] = J_cuda[index_n]; 
  south[ty][tx] = J_cuda[index_s];
  if ( by == 0 ){
  north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx]; 
  }
  else if ( by == gridDim.y - 1 ){
  south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
  }
   __syncthreads();
 
  west[ty][tx] = J_cuda[index_w];
  east[ty][tx] = J_cuda[index_e];

  if ( bx == 0 ){
  west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty]; 
  }
  else if ( bx == gridDim.x - 1 ){
  east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
  }
 
  __syncthreads();
  
 

  temp[ty][tx]      = J_cuda[index];

  __syncthreads();

   jc = temp[ty][tx];

   if ( ty == 0 && tx == 0 ){ //nw
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }	    
   else if ( ty == 0 && tx == BLOCK_SIZE-1 ){ //ne
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx]  - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == 0 ){//sw
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }

   else if ( ty == 0 ){ //n
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == BLOCK_SIZE -1 ){ //e
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1){ //s
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == 0 ){ //w
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else{  //the data elements which are not on the borders 
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }


    g2 = ( n * n + s * s + w * w + e * e ) / (jc * jc);

    l = ( n + s + w + e ) / jc;

	num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
	den  = 1 + (.25*l);
	qsqr = num/(den*den);

	// diffusion coefficent (equ 33)
	den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
	c = 1.0 / (1.0+den) ;

    // saturate diffusion coefficent
	if (c < 0){temp_result[ty][tx] = 0;}
	else if (c > 1) {temp_result[ty][tx] = 1;}
	else {temp_result[ty][tx] = c;}

    __syncthreads();

    C_cuda[index] = temp_result[ty][tx];
	E_C[index] = e;
	W_C[index] = w;
	S_C[index] = s;
	N_C[index] = n;

}
