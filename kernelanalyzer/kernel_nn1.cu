#include "hip/hip_runtime.h"
__global__ void executeFirstLayer(float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU)
{
	int blockID=blockIdx.x;
	int pixelX=threadIdx.x;
	int pixelY=threadIdx.y;


	int weightBegin=blockID*26;
	int windowX=pixelX*2;
	int windowY=pixelY*2;

	float result=0;

	result+=Layer1_Weights_GPU[weightBegin];

	++weightBegin;

	for(int i=0;i<25;++i)
	{
		result+=Layer1_Neurons_GPU[(windowY*29+windowX+kernelTemplate[i])+(29*29*blockIdx.y)]*Layer1_Weights_GPU[weightBegin+i];
	}

	result=(1.7159*tanhf(0.66666667*result));

	Layer2_Neurons_GPU[(13*13*blockID+pixelY*13+pixelX)+(13*13*6*blockIdx.y)]=result;

}


