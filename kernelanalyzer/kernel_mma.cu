
#include <hip/hip_runtime.h>
__global__
void kernel(float* dA, float* dB, float* dC, int dim)
{
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int idy = threadIdx.y + blockIdx.y*blockDim.y;
    if(idx<dim && idy<dim)
        dC[idx+idy*dim]=dA[idx+idy*dim]+dB[idx+idy*dim];
}
